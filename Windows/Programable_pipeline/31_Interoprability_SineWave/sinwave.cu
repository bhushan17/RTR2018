
#include <hip/hip_runtime.h>


__global__ void sinwave_vbo_kernel(float4* pos, unsigned int width, unsigned int height, float animtype)
{

	unsigned int Y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int X = blockIdx.x * blockDim.x + threadIdx.x;
	
	
	
	float u = X/float(width);
	float v = Y/float(height);
	
	u = ( u * 5.0) - 3.0;
	v = (v * 5.0) - 3.0;
	
	float frequency = 0.5f;
	
	float w = sinf(frequency * u + animtype) * cosf(frequency * v + animtype) ;
	          
	
	pos[Y * width + X] = make_float4(u,w,v,1.0);
	
}


void launchCudaKernel(float4 *pPos, int meshWidth, int meshHeight, float type)
{
 
  dim3 DimBlock=dim3(8,8,1);
  dim3 DimGrid=dim3(meshWidth / DimBlock.x, meshHeight/DimBlock.y, 1);
  
  sinwave_vbo_kernel<<<DimGrid, DimBlock>>>(pPos, meshWidth, meshHeight, type);
  
}


  
	